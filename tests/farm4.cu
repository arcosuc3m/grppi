/**
* @version		GrPPI v0.1
* @copyright		Copyright (C) 2017 Universidad Carlos III de Madrid. All rights reserved.
* @license		GNU/GPL, see LICENSE.txt
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You have received a copy of the GNU General Public License in LICENSE.txt
* also available in <http://www.gnu.org/licenses/gpl.html>.
*
* See COPYRIGHT.txt for copyright notices and details.
*/
#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>
#include <ppi/farm.h>



using namespace std;
using namespace grppi;

void farm_example1() {

    int a = 20000;

    auto p = parallel_execution_thrust(1, thrust::cuda::par);
    hipGetDeviceCount(&(p.num_gpus));

    farm(p,
        // Farm generator as lambda
        [&]() {
            a--; 
            if ( a == 0 ) { 
                return optional< vector<int> >();
            } 
            else {
                return optional< vector<int> >( vector<int>(1000,a) );
            }
        },
        // Farm kernel as lambda
	[] __device__ (int in)->int
           {
               return in;
           },
        [&](vector<int> v){
           std::cout<<v[0]<<"\n";
        }

 
    );
}

int main() {

    //$ auto start = std::chrono::high_resolution_clock::now();
    farm_example1();
    //$ auto elapsed = std::chrono::high_resolution_clock::now() - start;

    //$ long long microseconds = std::chrono::duration_cast<std::chrono::microseconds>( elapsed ).count();
    //$ std::cout << "Execution time : " << microseconds << " us" << std::endl;

    return 0;
}
