/**
* @version		GrPPI v0.1
* @copyright		Copyright (C) 2017 Universidad Carlos III de Madrid. All rights reserved.
* @license		GNU/GPL, see LICENSE.txt
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You have received a copy of the GNU General Public License in LICENSE.txt
* also available in <http://www.gnu.org/licenses/gpl.html>.
*
* See COPYRIGHT.txt for copyright notices and details.
*/
#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>
#include <ppi/reduce.h>

using namespace std;
using namespace grppi;

void map_example1() {
    std::vector<int> in(10000000);
    for(int i=0;i<in.size();i++) in[i] = 1;
    int out;
  
    auto p = parallel_execution_thrust(1, thrust::cuda::par);
    hipGetDeviceCount(&(p.num_gpus));

    reduce(p, in.begin(), in.end(), out, [] __device__ (int & in, int & out){ return in + out; });
    std::cout<<"REDUCE : "<< out <<std::endl;
}

int main() {

    //$ auto start = std::chrono::high_resolution_clock::now();
    map_example1();
    //$ auto elapsed = std::chrono::high_resolution_clock::now() - start;

    //$ long long microseconds = std::chrono::duration_cast<std::chrono::microseconds>( elapsed ).count();
    //$ std::cout << "Execution time : " << microseconds << " us" << std::endl;

    return 0;
}
