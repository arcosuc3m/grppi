/**
* @version		GrPPI v0.1
* @copyright		Copyright (C) 2017 Universidad Carlos III de Madrid. All rights reserved.
* @license		GNU/GPL, see LICENSE.txt
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You have received a copy of the GNU General Public License in LICENSE.txt
* also available in <http://www.gnu.org/licenses/gpl.html>.
*
* See COPYRIGHT.txt for copyright notices and details.
*/
#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>
#include <ppi/mapreduce.h>
#include <gtest/gtest.h>

using namespace std;
using namespace grppi;

int mapreduce_example1() {
    std::vector<std::vector<int>> mat(10000);
    for(int i=0;i<mat.size();i++) {
        mat[i] = std::vector<int> (10000);
        for(int j=0;j<mat[i].size();j++){
            mat[i][j] = 1;
        }
    }
    std::vector<int> v(10000);
    for( int i= 0 ; i< v.size(); i++){
         v[i] = 2;
    }
    std::vector<int> out(10000);

    auto p = parallel_execution_thrust(1, thrust::cuda::par);
    hipGetDeviceCount(&(p.num_gpus));

    int result = 0;
    int val = 3;

    map_reduce(p, v.begin(), v.end(),
           [=] __device__  (int in){ 
                  return val * in;  
           },
           [] __device__ (int in, int out){
		 return out + in; 
           },
           result
   );

   return result;

//    for(int i = 0; i< out.size();i++) std::cout<<"REDUCTION ["<<i<<"] = "<<out[i]<<std::endl;
}

TEST(GrPPI, mapreduce_gpu ){
    EXPECT_EQ(60000, mapreduce_example1() );
}

int main(int argc, char **argv) {
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
